#include "AESFile.cuh"
#include "AESCore.cuh"
#include "AES.cuh"


bool EncryptFile(const std::string& inFile, const std::string& outFile, unsigned char* key, enum keySize size) {
    const int blockSize = 16;  // AES block size is 128 bits (16 bytes)
    std::ifstream input(inFile, std::ios::binary);
    std::ofstream output(outFile, std::ios::binary);

    if (!input.is_open() || !output.is_open()) {
        std::cerr << "Failed to open files!" << std::endl;
        return false;
    }

    // Determine the size of the file and the number of blocks
    input.seekg(0, std::ios::end);
    size_t fileSize = input.tellg();
    input.seekg(0, std::ios::beg);
    int numBlocks = (fileSize + blockSize - 1) / blockSize;  // Rounds up

    // Allocate host memory for input and output
    unsigned char* h_input = new unsigned char[numBlocks * blockSize];
    unsigned char* h_output = new unsigned char[numBlocks * blockSize];

    // Read the input file into host memory
    input.read(reinterpret_cast<char*>(h_input), fileSize);
    input.close();

    // Padding the last block if necessary
    if (fileSize % blockSize != 0) {
        std::memset(h_input + fileSize, 0, numBlocks * blockSize - fileSize);
    }

    // Set the number of rounds based on key size
    int nbrRounds;
    switch (size) {
    case SIZE_16: nbrRounds = 10; break;
    case SIZE_24: nbrRounds = 12; break;
    case SIZE_32: nbrRounds = 14; break;
    default: return false;
    }

    // Allocate device memory
    unsigned char* d_input, * d_output, * d_expandedKey;
    hipMalloc(&d_input, numBlocks * blockSize * sizeof(unsigned char));
    hipMalloc(&d_output, numBlocks * blockSize * sizeof(unsigned char));
    hipMalloc(&d_expandedKey, (nbrRounds + 1) * blockSize * sizeof(unsigned char));

    // Copy input data and expanded key to device
    hipMemcpy(d_input, h_input, numBlocks * blockSize * sizeof(unsigned char), hipMemcpyHostToDevice);

    unsigned char expandedKey[240];  // Maximum size for AES-256
    CreateExpandKey(expandedKey, key, size, (nbrRounds + 1) * blockSize);
    hipMemcpy(d_expandedKey, expandedKey, (nbrRounds + 1) * blockSize * sizeof(unsigned char), hipMemcpyHostToDevice);

    // Launch AES kernel
    int threadsPerBlock = 256;
    int blocksPerGrid = (numBlocks + threadsPerBlock - 1) / threadsPerBlock;
    AES_EncryptKernel << <blocksPerGrid, threadsPerBlock >> > (d_input, d_output, d_expandedKey, numBlocks, nbrRounds);

    // Copy the encrypted output back to host
    hipMemcpy(h_output, d_output, numBlocks * blockSize * sizeof(unsigned char), hipMemcpyDeviceToHost);

    // Write the output file
    output.write(reinterpret_cast<char*>(h_output), numBlocks * blockSize);
    output.close();

    // Free memory
    hipFree(d_input);
    hipFree(d_output);
    hipFree(d_expandedKey);
    delete[] h_input;
    delete[] h_output;

    return true;
}

bool DecryptFile(const std::string& inFile, const std::string& outFile, unsigned char* key, enum keySize size) {
    const int blockSize = 16;  // AES block size is 128 bits (16 bytes)
    std::ifstream input(inFile, std::ios::binary);
    std::ofstream output(outFile, std::ios::binary);

    if (!input.is_open() || !output.is_open()) {
        std::cerr << "Failed to open files!" << std::endl;
        return false;
    }

    // Determine the size of the file and the number of blocks
    input.seekg(0, std::ios::end);
    size_t fileSize = input.tellg();
    input.seekg(0, std::ios::beg);
    int numBlocks = (fileSize + blockSize - 1) / blockSize;  // Rounds up

    // Allocate host memory for input and output
    unsigned char* h_input = new unsigned char[numBlocks * blockSize];
    unsigned char* h_output = new unsigned char[numBlocks * blockSize];

    // Read the input file into host memory
    input.read(reinterpret_cast<char*>(h_input), fileSize);
    input.close();

    // Set the number of rounds based on key size
    int nbrRounds;
    switch (size) {
    case SIZE_16: nbrRounds = 10; break;
    case SIZE_24: nbrRounds = 12; break;
    case SIZE_32: nbrRounds = 14; break;
    default: return false;
    }

    // Allocate device memory
    unsigned char* d_input, * d_output, * d_expandedKey;
    hipMalloc(&d_input, numBlocks * blockSize * sizeof(unsigned char));
    hipMalloc(&d_output, numBlocks * blockSize * sizeof(unsigned char));
    hipMalloc(&d_expandedKey, (nbrRounds + 1) * blockSize * sizeof(unsigned char));

    // Copy input data and expanded key to device
    hipMemcpy(d_input, h_input, numBlocks * blockSize * sizeof(unsigned char), hipMemcpyHostToDevice);

    unsigned char expandedKey[240];  // Maximum size for AES-256
    CreateExpandKey(expandedKey, key, size, (nbrRounds + 1) * blockSize);
    hipMemcpy(d_expandedKey, expandedKey, (nbrRounds + 1) * blockSize * sizeof(unsigned char), hipMemcpyHostToDevice);

    // Launch AES decryption kernel
    int threadsPerBlock = 256;
    int blocksPerGrid = (numBlocks + threadsPerBlock - 1) / threadsPerBlock;
    AES_DecryptKernel << <blocksPerGrid, threadsPerBlock >> > (d_input, d_output, d_expandedKey, numBlocks, nbrRounds);

    // Copy the decrypted output back to host
    hipMemcpy(h_output, d_output, numBlocks * blockSize * sizeof(unsigned char), hipMemcpyDeviceToHost);

    // Write the output file
    output.write(reinterpret_cast<char*>(h_output), numBlocks * blockSize);
    output.close();

    // Free memory
    hipFree(d_input);
    hipFree(d_output);
    hipFree(d_expandedKey);
    delete[] h_input;
    delete[] h_output;

    return true;
}


