﻿// main.cpp
#include "AES.cuh"

#include <chrono>
#include <cstdlib>

void DisplayAESExplanationkey(unsigned char* key, enum keySize size)
{
    const int expandedKeySizeDisplay = 240;

    // the expanded key
    unsigned char expandedKeyDisplay[expandedKeySizeDisplay];


    CreateExpandKey(expandedKeyDisplay, key, size, expandedKeySizeDisplay);

    std::cout << "Expanded Key:\n";
    for (int i = 0; i < expandedKeySizeDisplay; i++) {
        // Print the block number at the beginning of each new line
        if (i % 16 == 0) {
            std::cout << (i / 16 + 1) << ": ";  // Block number starts from 1
        }

        std::cout << std::hex << std::setw(2) << std::setfill('0')
            << static_cast<int>(expandedKeyDisplay[i]);

        // Insert a space after each byte for readability
        if ((i + 1) % 16 == 0) {
            std::cout << std::endl;  // Print a newline after every 16 bytes
        }
        else {
            std::cout << " ";  // Print a space otherwise
        }
    }


    printf("\nExpanded Key (HEX format):\n");

    for (int i = 0; i < expandedKeySizeDisplay; i++)
    {
        printf("%2.2x%c", expandedKeyDisplay[i], ((i + 1) % 16) ? ' ' : '\n');
    }



}

#pragma region CUDAFILEVersion
int main(int argc, char* argv[])
{
    // Define file paths for input and output
    std::string inputFilePath = "FileToEncrypt/10mb.txt";           // Input file to encrypt
    std::string encryptedFilePath = "EncryptFile/10mbEncrypted.bin"; // Encrypted output file
    std::string decryptedFilePath = "DecryptFile/10mb_unencrypted.txt"; // Decrypted output file

    // Define the key and key size (256-bit in this example)
    unsigned char key[32] = { "HelloWorldThisIsAKey12345678" }; // Example key
    enum keySize size = SIZE_32;  // You can change this to SIZE_16 or SIZE_24 for 128-bit or 192-bit keys

    // Display the expanded key for informational purposes
    DisplayAESExplanationkey(key, size);

    // Measure encryption time
    auto encryptionStart = std::chrono::high_resolution_clock::now();

    // Encrypt the file
    if (EncryptFile(inputFilePath, encryptedFilePath, key, size)) {
        std::cout << "File encryption completed successfully!" << std::endl;
    }
    else {
        std::cerr << "File encryption failed!" << std::endl;
        return -1;
    }

    auto encryptionEnd = std::chrono::high_resolution_clock::now();
    std::chrono::duration<double> encryptionDuration = encryptionEnd - encryptionStart;
    std::cout << "Time taken for encryption: " << encryptionDuration.count() << " seconds" << std::endl;

    // Measure decryption time
    auto decryptionStart = std::chrono::high_resolution_clock::now();

    // Decrypt the file
    if (DecryptFile(encryptedFilePath, decryptedFilePath, key, size)) {
        std::cout << "File decryption completed successfully!" << std::endl;
    }
    else {
        std::cerr << "File decryption failed!" << std::endl;
        return -1;
    }

    auto decryptionEnd = std::chrono::high_resolution_clock::now();
    std::chrono::duration<double> decryptionDuration = decryptionEnd - decryptionStart;
    std::cout << "Time taken for decryption: " << decryptionDuration.count() << " seconds" << std::endl;

    // Optionally, you can open the original and decrypted files for comparison
    std::string openOriginalFileCommand = "start " + inputFilePath;
    system(openOriginalFileCommand.c_str());

    std::string openDecryptedFileCommand = "start " + decryptedFilePath;
    system(openDecryptedFileCommand.c_str());

    return 0;
}
#pragma endregion

#pragma region AESStuff
//
//void DisplayAESExplanationkey(unsigned char* key, enum keySize size)
//{
//    const int expandedKeySizeDisplay = 240;
//
//    // the expanded key
//    unsigned char expandedKeyDisplay[expandedKeySizeDisplay];
//
//
//    CreateExpandKey(expandedKeyDisplay, key, size, expandedKeySizeDisplay);
//
//    std::cout << "Expanded Key:\n";
//    for (int i = 0; i < expandedKeySizeDisplay; i++) {
//        // Print the block number at the beginning of each new line
//        if (i % 16 == 0) {
//            std::cout << (i / 16 + 1) << ": ";  // Block number starts from 1
//        }
//
//        std::cout << std::hex << std::setw(2) << std::setfill('0')
//            << static_cast<int>(expandedKeyDisplay[i]);
//
//        // Insert a space after each byte for readability
//        if ((i + 1) % 16 == 0) {
//            std::cout << std::endl;  // Print a newline after every 16 bytes
//        }
//        else {
//            std::cout << " ";  // Print a space otherwise
//        }
//    }
//
//
//
//}
//
//void FileAES()
//{
//
//
//    unsigned char key[32] = { "Hallo World" };
//    enum keySize size = SIZE_32;
//
//
//    DisplayAESExplanationkey(key, size);
//
//    std::string inFile = "FileToEncrypt/10mb.txt";
//    std::string outFile = "EncryptFile/10mbEncrypted.bin";
//
//
//
//    std::ifstream encryptInput(inFile, std::ios::binary);
//    if (!encryptInput.is_open()) {
//        std::cerr << "Failed to open input file: " << inFile << std::endl;
//        return;
//    }
//    else {
//        std::cout << "Input file opened successfully: " << inFile << std::endl;
//    }
//
//
//    std::ofstream encrypOutput(outFile, std::ios::binary);
//    if (!encrypOutput.is_open()) {
//        std::cerr << "Failed to open output file: " << outFile << std::endl;
//        return;
//    }
//    else {
//        std::cout << "Output file created/opened successfully: " << outFile << std::endl;
//    }
//
//
//    encryptInput.close();
//    encrypOutput.close();
//
//    auto start = std::chrono::high_resolution_clock::now();
//
//    // Encrypt the file
//    if (EncryptFile(inFile, outFile, key, size)) {
//        std::cout << "File encryption completed successfully!" << std::endl;
//    }
//    else {
//        std::cerr << "File encryption failed!" << std::endl;
//    }
//
//    auto end = std::chrono::high_resolution_clock::now();
//
//    // Calculate the duration
//    std::chrono::duration<double> duration = end - start;
//
//    // Output the duration
//    std::cout << "Time taken for encryption: " << duration.count() << " seconds" << std::endl;
//
//
//    std::string DecryptOutputFile = "DecryptFile/10mb_unencrypted.txt";
//
//    std::ifstream DecryptInput(outFile, std::ios::binary);
//    if (!DecryptInput.is_open()) {
//        std::cerr << "Failed to open input file: " << outFile << std::endl;
//        return;
//    }
//    else {
//        std::cout << "Input file opened successfully: " << outFile << std::endl;
//    }
//
//
//    std::ofstream DecryptOutput(DecryptOutputFile, std::ios::binary);
//    if (!DecryptOutput.is_open()) {
//        std::cerr << "Failed to open output file: " << DecryptOutputFile << std::endl;
//        return;
//    }
//    else {
//        std::cout << "Output file created/opened successfully: " << DecryptOutputFile << std::endl;
//    }
//
//
//    DecryptInput.close();
//    DecryptOutput.close();
//
//
//    auto DecryptStart = std::chrono::high_resolution_clock::now();
//
//    // Encrypt the file
//    if (DecryptFile(outFile, DecryptOutputFile, key, size)) {
//        std::cout << "File encryption completed successfully!" << std::endl;
//    }
//    else {
//        std::cerr << "File encryption failed!" << std::endl;
//    }
//
//    auto DecryptEnd = std::chrono::high_resolution_clock::now();
//
//    // Calculate the duration
//    std::chrono::duration<double> decryptDuration = DecryptEnd - DecryptStart;
//
//    // Output the duration
//    std::cout << "Time taken for Decryption: " << decryptDuration.count() << " seconds" << std::endl;
//
//    std::string openOriginalFileCommand = "start " + inFile;
//    system(openOriginalFileCommand.c_str());
//
//    std::string openDecryptedFileCommand = "start " + DecryptOutputFile;
//    system(openDecryptedFileCommand.c_str());
//
//}
//int main(int argc, char* argv[])
//{
//    FileAES();
//    return 0;
//}
#pragma endregion

#pragma region CUDASmallAES
//int main() {
//    // Define a small input message (exactly 16 bytes, no padding required)
//    const char* testInput = "abcdef1234567890";  // 16 bytes (exactly one AES block)
//    unsigned char encrypted[16];
//    unsigned char decrypted[16];
//    
//    // AES key (this should match the key size you're using in the real test)
//    unsigned char key[32] = { "Hallo World" };  // Example key, AES-256 (32 bytes)
//
//    // Print the original message
//    std::cout << "Original message: " << testInput << std::endl;
//
//    // Encrypt the message
//    char resultEncrypt = AES_Encrypt((unsigned char*)testInput, encrypted, key, SIZE_32);
//    if (resultEncrypt != SUCCESS) {
//        std::cerr << "Encryption failed with error code: " << resultEncrypt << std::endl;
//        return -1;
//    }
//
//    // Print encrypted message in hex
//    std::cout << "Encrypted message (hex): ";
//    for (int i = 0; i < 16; ++i) {
//        std::cout << std::hex << (int)encrypted[i] << " ";
//    }
//    std::cout << std::endl;
//
//    // Decrypt the message
//    char resultDecrypt = AES_Decrypt(encrypted, decrypted, key, SIZE_32);
//    if (resultDecrypt != SUCCESS) {
//        std::cerr << "Decryption failed with error code: " << resultDecrypt << std::endl;
//        return -1;
//    }
//
//    // Print the decrypted message
//    std::cout << "Decrypted message: " << decrypted << std::endl;
//
//    // Compare original and decrypted message
//    if (std::memcmp(testInput, decrypted, 16) == 0) {
//        std::cout << "Decryption successful!" << std::endl;
//    } else {
//        std::cout << "Decryption failed. Original and decrypted message do not match." << std::endl;
//    }
//
//    return 0;
//}
#pragma endregion



